#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <lib.cuh>

__global__ void kernel(int* a, int* b) {
	a[threadIdx.x] += b[threadIdx.x];
}

int kernelWrapper(int* a, int* b, const unsigned int length) {
	checkCudaErrors(hipSetDevice(0));
	int* ad;
	int* bd;
	const unsigned int intSize = length * sizeof(int);
	
	checkCudaErrors(hipMalloc((void**)&ad, intSize));
	checkCudaErrors(hipMalloc((void**)&bd, intSize));
	checkCudaErrors(hipMemcpy(ad, a, intSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(bd, b, intSize, hipMemcpyHostToDevice));
	
	dim3 dimBlock(length, 1);
	dim3 dimGrid(1, 1);
	kernel<<<dimGrid, dimBlock>>>(ad, bd);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize()); 
	checkCudaErrors(hipMemcpy(a, ad, intSize, hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(ad));
	checkCudaErrors(hipFree(bd));
	printf("%s\n", a);
	return 0;
}